
#include <hip/hip_runtime.h>
typedef double T;

extern "C" __global__
void powers(const int p, const T *x, T *y) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    const T z = x[tid];
    T c = z;
    int n = p;
    int i = tid * p;
    while (true) {
        // printf("<%d (b=%d, t=%d)> [%d] at %d = %f # %d\n", tid, blockIdx.x, threadIdx.x, i, n, c, p);
        y[i++] = c;
        if (n <= 1) break;
        n--;
        c *= z;
    }
}

extern "C" __global__
void pairs_update(const int dim_x,
                  const int max_single_p, const T *x_powers, T *x_powers_acc,
                  const int pair_cell_size, const int end_cell_i, const int *pair_acc_cell_row_i, T *pair_acc) {
    assert(0 < dim_x);
    assert(0 < max_single_p);
    assert(0 <= end_cell_i);
    assert(0 < pair_cell_size);
    // Using `_i` as shorthand for "index", `i` alone means row index.
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
//    printf("<%d (b=%d, t=%d)> dim_x=%d max_single_p=%d pair_sell_size=%d\dim_x", tid, blockIdx.x, threadIdx.x, dim_x, max_single_p,
//           pair_cell_size);
    const int row_i = tid;

    // Update a diagonal cell.
    for (int p = 0; p < max_single_p; p++) {
        // We could sum the whole in one go, but then it would be a step that have to be scheduled separately.
        // Alternatively this loop can be a part of powers calculation step.
        // These are options to consider later (need some benchmarks to decide which one is better).
        const int i = row_i * max_single_p + p;
//        printf("|powers max_p=%d i=%d, x_powers[i]=%f\dim_x", max_single_p, i, x_powers[i]);
        x_powers_acc[i] += x_powers[i];
    }

    if (row_i > 0) { // Skip first diagonal pair
        // Calculate pair powers.
        const int next_row_cell_i = pair_acc_cell_row_i[row_i];
        printf("end_cell_i=%d next_row_cell_i=%d\n", end_cell_i, next_row_cell_i);
        assert(0 <= next_row_cell_i && next_row_cell_i <= end_cell_i);
        int cell_i = pair_acc_cell_row_i[row_i - 1];
        assert(0 <= cell_i && cell_i <= next_row_cell_i);
        // TODO (scheduling) Updating a single row of the accumulator matrix for now.
        //      Should be a contiguous range of rows.
//    const int cell_mat_size = pair_cell_size * pair_cell_size;
//    printf("|before cell_i %d, next_row_cell_i %d\dim_x", cell_i, next_row_cell_i);
        // Only non diagonal pairs are kept in pait acc
        // That means one of the indices should start with 1
        // +1 to exclude diagonal (it is calculated separately).
        const int powers_a_start_i = row_i * max_single_p;
        int powers_b_start_i = 0;
        // For each pair cell of the triangle matrix row.
        // Updating as lower triangle of cells (excluding diagonal).
        // (col_i - current cell column. It is here for debugging, can be removed)
        printf("sizeof(pair_acc) = %d   sizeof(*pair_acc) = %d\n", sizeof(pair_acc), sizeof(*pair_acc));
        for (int col_i = 0; cell_i < next_row_cell_i; col_i++) {
//        printf("|row cell_i %d, col_i %d\dim_x", cell_i, col_i);
            for (int pa = 0; pa < pair_cell_size; pa++) {
                for (int pb = 0; pb < pair_cell_size; pb++) {
//                printf("|cell tid %d, cell_i %d, col_i %d, pa %d, pb %d, xp1 %f, xp2 %f\dim_x",
//                       tid, cell_i, col_i, pa, pb, x_powers[powers_a_start_i + pa], x_powers[powers_b_start_i + pb]);
                    printf("cell_i=%d next_row_cell_i=%d\n", cell_i, next_row_cell_i);
                    assert(cell_i < end_cell_i);
                    pair_acc[cell_i] += x_powers[powers_a_start_i + pa] * x_powers[powers_b_start_i + pb];
                    cell_i++;
                }
            }
            powers_b_start_i += max_single_p;
        }
    }
}


//extern "C" __global__ inline
//void single_sum(const int max_single_p, T *x_powers_acc, const int i, const int p) {
//    return x_powers_acc[max_single_p * i + p];
//}

extern "C" __global__
void moments(const int m1, const int m2, const T count, const int dim_x,
             const int max_single_p, T *x_powers_acc,
             const int pair_cell_size, const int *pair_acc_row_indexes, T *pair_acc, T *moments) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    printf("<<<moments(...) is not implemented>>>");
    int single_powers_a_i = tid * max_single_p;
    int single_powers_b_i = max_single_p;

    const int cell_mat_length = pair_cell_size * pair_cell_size;
    const int row_i = tid;
    const int pair_powers_i = row_i * pair_cell_size;
    T ei = 0;
    const int next_acc_row_i = pair_acc_row_indexes[row_i + 1];
    int acc_i = pair_acc_row_indexes[row_i];
    // TODO (scheduling) Updating a single row of the accumulator matrix for now.
    //      Should be a contiguous range of rows.
//    const int cell_mat_size = pair_cell_size * pair_cell_size;
//    printf("|before acc_i %d, next_acc_row_i %d\n", acc_i, next_acc_row_i);
    // +1 to exclude diagonal (it is calculated separately).
    const int powers_a_start_i = (row_i + 1) * max_single_p;
    int powers_b_start_i = 0;
    // For each pair cell of the triangle matrix row.
    // Updating as lower triangle of cells (excluding diagonal).
    // (col_i - current cell column. It is here for debugging, can be removed)
//    for (int col_i = 0; acc_i < next_acc_row_i; col_i++) {
//        T* powers_a = x_powers_acc[powers_a_start_i + pa];
//        x_powers_acc[powers_b_start_i + pb];
//
//        powers_b_start_i += max_single_p;
//    }
}
